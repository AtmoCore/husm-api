#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

__global__ void husm_kernel(float* input, int* indices, float* output, int num_streams) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_streams) return;

    int start = indices[idx];
    int end = indices[idx + 1];

    float sum = 0.0f;
    for (int i = start; i < end; ++i) {
        sum += input[i];
    }

    output[idx] = sum;
}

extern "C" void launch_husm(const std::vector<std::vector<float>>& streams, std::vector<float>& result) {
    int total_elements = 0;
    int num_streams = streams.size();

    std::vector<float> flat_input;
    std::vector<int> indices;

    indices.push_back(0);
    for (const auto& stream : streams) {
        total_elements += stream.size();
        flat_input.insert(flat_input.end(), stream.begin(), stream.end());
        indices.push_back(total_elements);
    }

    float *d_input, *d_output;
    int *d_indices;

    hipMalloc(&d_input, flat_input.size() * sizeof(float));
    hipMalloc(&d_indices, indices.size() * sizeof(int));
    hipMalloc(&d_output, num_streams * sizeof(float));

    hipMemcpy(d_input, flat_input.data(), flat_input.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_indices, indices.data(), indices.size() * sizeof(int), hipMemcpyHostToDevice);

    int threads_per_block = 256;
    int blocks = (num_streams + threads_per_block - 1) / threads_per_block;

    husm_kernel<<<blocks, threads_per_block>>>(d_input, d_indices, d_output, num_streams);

    result.resize(num_streams);
    hipMemcpy(result.data(), d_output, num_streams * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_indices);
    hipFree(d_output);
}
